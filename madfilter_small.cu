/* MAD Filter on GPU

Version 1.0
Runs on single bin size
Input:  filename
        Number of samples to filter
        Bin size
        Threshold (multiple of sigma)
        Option for filtering
 
Basic version using histogram method for median. 24/01/13
Replace with random numbers. Added 28/01/13
Finding mean and rms before and after filtering. Added 02/02/13 
Copy back only flags file (bool)

Compile it with following line:
nvcc -Xptxas="-v" -o madfilter_small madfilter_small.cu -arch=sm_20 

(Rohini Joshi, 2013 - rmjoshi.06@gmail.com)
*/
  
#include<hip/hip_runtime.h>
#include<hiprand/hiprand.h>  		// random num generation in cuda
#include<hiprand/hiprand_kernel.h>       // random num generation in cuda
#include<stdio.h>
#include<sys/time.h>
#include<iostream>

using std::cerr;
using std::cout;
using std::endl;
 
// __device__ --> is a dev fn to be run on GRID and can be called only from kernel or device fn

__device__ float randomnumber(int t, int i){

hiprandState s; 
float x;
// curand_init() sets up an initial state s. with seed t(thread id) and sequence number 0 and offset i
// Each bin is filtered with a separate thread. Thus normal distribution of random numbers is preserved within a bin
hiprand_init(t, 0, i, &s);  // t is a seed, i is offset in seq of random numbers 
// Generate random number from normal distribution
x = hiprand_normal(&s);

return x;
}

__global__ void madfilter( int *d_data, int binsize, int bins, int op, float *dev, int *not_flagged_data, bool *d_flag, float *d_rms_b, float *d_rms_a, float *d_mad, int mult){
    
// {0} initialised the whole array. blockDim = number of threads/block=32, tid indexes all threads in the grid
// everything below runs for each thread through threadIdx.x and blockIdx.x
int i, j=0, c=0,d,flag=0,odd=0,sum=0, sumsq=0,histdev[256] = {0},hist[256] = {0}, tid = threadIdx.x + blockIdx.x * blockDim.x;
int lw = tid * (binsize);   // the index in the original data array for each bin beginning
int up = lw + (binsize);    //                    same                           end
float mean, med, mad, thresh;

// variable j is to store effective size of bin (after flagging extremities)
if (tid < bins){    // end crap is not accessed

/* Flagging and generating histogram */
for ( i=lw; i<up; i++){
	sum += d_data[i];
	sumsq += d_data[i]*d_data[i];

	// Flag extremities
        if((d_data[i]==-128) || (d_data[i] == 127)){
                continue;
        }else{
                //atomicAdd( &hist[h+d_data[i]+128], 1 );
                hist[d_data[i] + 128] += 1;
                not_flagged_data[lw+j] = d_data[i];
		j+=1;
        }
}
/* Find RMS before filtering */
mean = sum/(binsize);
d_rms_b[tid] = sqrtf( sumsq/(binsize) - mean*mean );
sum = 0;sumsq = 0;
/* Find median. Two methods for even/odd sizes. Modify if data is 4 bit
flag = 1/0 if median is floating point/int
odd = 1/0 if data set is odd/even 
median can only be float if data set is even */

if (j%2 == 0){
        d = j/2;
        for ( i=0; i<(256); i++){
                c = c + hist[i];
                if (c==d){
                        med =(float)( (2*(i) + 1)*0.5 - 128 );
                        flag = 1;
                        break;
                }else if (c>d){
                        med = (i - 128);
                        break;
                }else
                        continue;
        }
}else{
        d = (j + 1)/2;
        odd = 1;
        c = 0;
        for ( i=0; i<(256); i++){
                c = c + hist[i];
                if (c >= d){
                        med = i - 128;
                        break;
                }
        }
}
//  MAD
int s = 0, ii;
if (flag == 0){
        for ( i=lw; i<lw+j; i++){
                dev[i] = fabs( not_flagged_data[i] - med );
                ii = (int)(ceil(dev[i]));
		histdev[ii] += 1;
        }
        /* two submethods for even/odd data sets */
        if (odd == 0){
        for ( i=0; i<(256); i++){
                s = s+histdev[i];
                if (s == d){
                        mad = (float)( (2*(i) + 1)*0.5 );
                        break;
                }else if (s > d ){
                        mad = i;
                        break;
                }else
                        continue;
        }
        }else{
        for ( i=0; i<(256); i++){
                s = s + histdev[i];
                if(s >= d){
                        mad = i;
                        break;
                }
        }
        }
}else{
        int p;
        for ( i=lw; i<lw+j; i++){
                dev[i] = (float)fabs( not_flagged_data[i] - med );
                p = (int) dev[i];
                histdev[p] += 1;
        }
        int s = 0;
        d = j/2;
        for ( i=0; i<(256); i++){
                s = s+histdev[i];
                if (s == d){
                        mad = (float)( (2*(i) + 1)*0.5 + 0.5 );
                        break;
                }else if (s > d){
                        mad = (float)( i + 0.5 );
                        break;
                }else
                        continue;
        }

}
  
d_mad[tid] = mad;
thresh = mult*1.4826*mad;
//filtering

// thresh = mult*1.48*mad
// if abs(d-med) > thresh ---> flag

for( i=lw; i<up; i++){
	if ( (fabsf(d_data[i]-med) > thresh) || (d_data[i] == -128) || (d_data[i] == 127)  ){
        	if(op == 0){
	                d_data[i] = 0;
	        }else if(op == 1){
        	        d_data[i] = med;
		}else if(op == 2){
			d_data[i] = rint(mean + 1.4826*mad*randomnumber(tid, i-lw));
	        }else if(op == 1){
			d_data[i] = thresh;
		}d_flag[i] = 0;
	}
	else{
		d_flag[i] = 1;
	}
	sum += d_data[i];
	sumsq += d_data[i]*d_data[i];
}

/* Find RMS after filtering */
mean = sum/(binsize);
d_rms_a[tid] = sqrtf( sumsq/(binsize) - mean*mean );

/*if(*op == 0){
	printf("replace with zero\n");
}else if(*op == 1){ 
	printf("replace with median\n");
}else if(*op == 2){    
         printf("replace with random number\n");
}
*/

}
}
  
int main(int argc, char *argv[]){

int i, mult_thresh, num, size, bsize, bins;
int *h_data, *d_data, *not_flagged_data, op_int;
float *h_rms_b, *h_rms_a, *d_rms_b, *d_rms_a, *h_mad, *d_mad, *dev;
double time1, time2;
FILE *fp;
char *fname, *ffname,*op;
struct timeval tim;
float time_initial_host, time_initial_dev, time_kernel, time_copyback;
bool *h_flag, *d_flag;

//Define and create CUDA events start and stop for timing GPU activity
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

// Parse input arguments
// filename	
fname = argv[1];
// Size of data to filter from the input file
size = atoi( argv[2] );
// Bin size to use
bsize = atoi( argv[3] );
// Multiple of MAD to use as threshold 
mult_thresh = atoi( argv[4] );
// Option to use for filtering (what to replace RFI with)
op = argv[5];

if (argc <= 5 ){
	system("./help_small.sh");
	exit(0);
}

// Number of whole bins that can be filtered in the dataset
bins = (int)size/bsize;
// size is now made a multiple of the bin size
size = bins*bsize;

gettimeofday(&tim, NULL);
time1 = tim.tv_sec + (tim.tv_usec/1000000.0);

/* Allocate and store input on host */
h_data = (int *)malloc(size*sizeof(int));	// actual data - will be read from SHM
h_rms_b = (float *)malloc(bins*sizeof(float));	// RMS before filtering for each bin    - for checking
h_rms_a = (float *)malloc(bins*sizeof(float));	// RMS after filtering for each bin    - for checking
h_mad = (float *)malloc(bins*sizeof(float));	// MAD value for each bin    - for checking
h_flag = (bool *)malloc(size*sizeof(bool));	// Flags 
ffname = (char *)malloc(30*sizeof(char));	// New file name in which filtered data will be written out - will become SHM
sprintf(ffname, "%s_filtered", fname);

// Store data in host memory from file   --- will change to reading from SHM continuously
fp = fopen(fname, "r");
if (fp == NULL){
        printf("Error in opening input file\n");
}
for(i=0; i<size; i++){
        fscanf(fp, "%d\n", &num);
        h_data[i] = num;
}
fclose(fp);

// As strcmp cannot be used in a kernel, convert the filtering option from char to integer here itself
if(!strcmp(op, "-z")){
	op_int=0;
}else if(!strcmp(op, "-m")){
	op_int=1;
}else if(!strcmp(op, "-r")){
	op_int=2;
}else if(!strcmp(op, "-c")){
	op_int=3;
}
gettimeofday(&tim, NULL);
time2 = tim.tv_sec + (tim.tv_usec/1000000.0);
time_initial_host = time2 - time1;

/* Allocate i/o and store input on device */
hipEventRecord( start, 0 ); // Start CUDA timer
hipMalloc( (void **)&d_data, size*sizeof(int) );
hipMalloc( (void **)&d_rms_b, bins*sizeof(float) );    // dont need this
hipMalloc( (void **)&d_rms_a, bins*sizeof(float) );    // dont need this
hipMalloc( (void **)&d_mad, bins*sizeof(float) );
hipMalloc( (void **)&dev, bins*sizeof(float)*bsize );
hipMalloc( (void **)&not_flagged_data, bins*sizeof(int)*bsize );
hipMalloc( (void **)&d_flag, size*sizeof(bool) );	// actual flags, can be bool
hipMemcpy( d_data, h_data, size*sizeof(int), hipMemcpyHostToDevice );

hipEventRecord( stop, 0 );	// Stop and store time elapsed
hipEventSynchronize(stop);
hipEventElapsedTime( &time_initial_dev, start, stop);

/* Setup grid and run kernel */
int blocks, threads=32;
blocks = (bins + threads - 1)/threads;	// Mathematically equivalent to a ceil(bins/threads) = number of blocks so that 1 thread/bin
printf("Grid dim [%d 1] Block dim [%d 1]\n", blocks, threads);

hipEventRecord(start, 0);   // to start timing

// send # of blocks and threads to the cuda kernel. dev is y-median(y), d_flag is array of bools, mult_thresh=3
// is asynchronous => comes back to cpu even before finishing
madfilter<<<blocks, threads>>>( d_data, bsize, bins, op_int, dev, not_flagged_data, d_flag, d_rms_b, d_rms_a, d_mad, mult_thresh );

hipEventRecord(stop, 0);
hipEventSynchronize(stop);   // makes sure gpu is done, and is part of the timing module. but can synchronise in other ways instead 
			      //if not insterested in timing
hipEventElapsedTime( &time_kernel, start, stop);

printf("Number of Bins = %d\n", bins);
printf("Time for executing kernel = %f msec\n", time_kernel);

/* Copy data back to host */
hipEventRecord(start, 0);

hipMemcpy( h_flag, d_flag, size*sizeof(bool), hipMemcpyDeviceToHost );
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime( &time_copyback, start, stop);
hipMemcpy( h_mad, d_mad, bins*sizeof(float), hipMemcpyDeviceToHost );
hipMemcpy( h_data, d_data, size*sizeof(int), hipMemcpyDeviceToHost );
hipMemcpy( h_rms_b, d_rms_b, bins*sizeof(float), hipMemcpyDeviceToHost );
hipMemcpy( h_rms_a, d_rms_a, bins*sizeof(float), hipMemcpyDeviceToHost );

// Free memory on the device
hipFree(d_data);
hipFree(d_mad);
hipFree(d_rms_b);
hipFree(d_rms_a);
hipFree(dev);
hipFree(not_flagged_data);
hipFree(d_flag);
hipEventDestroy(start); 
hipEventDestroy(stop); 
cerr << "Time to copyback = " << time_copyback << " ms" << endl;
cerr << "Total time = " << time_copyback + time_kernel << " ms" << endl;

// Write out to file
fp = fopen("mad.dat", "w");
if (fp == NULL){
        printf("Error in opening output file\n");
}
for(i=0; i<bins; i++){
        fprintf(fp, "%f\t%f\t%f\n", h_rms_b[i], h_rms_a[i], h_mad[i]);
}
fclose(fp);
fp = fopen(ffname, "w");
if (fp == NULL){
        printf("Error in opening output file\n");
}
for(i=0;i<size;i++){
	fprintf(fp, "%d\n", h_data[i]);
}
fclose(fp);
printf("Data copied back to host\n");
}
